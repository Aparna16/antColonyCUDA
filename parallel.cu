
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <fstream>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define Infinity 65536 /* pow (2, 16) */

/* 
	randdouble()

	Retorna um numero (double) aleatorio entre 0.0f e 1.0f. 
	
	Parametros: 	
	Saida:
		numero aleaorio entre 0.0 e 1.0.
*/
#define randdouble() ((double)rand()/(double)RAND_MAX) 

/*
	randomize()

	Atualiza o gerador de numeros pseudo-aletatorios. 

	Parametros: 
	Saida:		
*/

#define randomize() srand((unsigned)time(NULL))

/* 
	index()

	Mapeia uma posicao de uma matriz (2D) para um indice de um vetor (1D). 
	
	Parametros:
		length: numero de colunas da matriz
		line: indice da linha 
		column: indice da coluna 

	Saída:
		indice mapeado
*/ 
#define index(length,line,column) (column + line * length) 

using namespace std;

const int NUMBER_OF_ITERATIONS = 50;
const double INIT_PHEROMONE_AMOUNT = 1.0;
const double EVAPORATION_RATE = 0.5;
const double ALFA = 1; /* Influencia da trilha de feromonios */
const double BETA = 2; /* Influencia da informacao heuristica */

/* 
	load_instance()

	Inicializa uma instancia ( numero de cidades e matriz de distancias ) do TSP. 

	Parametros:
		filename: nome do arquivo 
		n_cities: numero de cidades ( passagem por referencia ) 

	Saida: 
		matriz de distancias ( distancias euclidianas )
*/
int *load_instance ( char const *filename, int &n_cities ); 

/* 
	calculate_pathcost()

	Calcula o custo (soma dos custos de todas as arestas) de um caminho . 

	Parametros: 
		distances: matriz de distancias 
		path: caminho ( solucao ) 
		n_cities: numero de cidades 

	Saida:
		custo ( soma de todas as distancias ) do caminho
*/
int calculate_pathcost ( int *distances, int *path, int n_cities ); 

/* 
	best_solution()

	Retorna a melhor entre as solucoes geradas. 

	Parametros: 
		ants: matriz de solucoes 
		distances: matriz de distancias ( entre cidades ) 
		n_ants: numero de formigas 
		n_cities: numero de cidades

	Saida:
		melhor solucao encontrada 
*/
int *best_solution ( int *tours, int *distances, int n_ants, int n_cities ); 

/* 
	evaporate()

	Atualiza a matriz de feromonios aplicando evaporacao. 
	Para cada vertice, multiplica-se a taxa de evaporacao ( EVAPORATION_RATE ). 

	Parametros: 
		pheromones: matriz de feromonios 
		n_cities: numero de cidades 

	Saida:
		matriz de feromonios atualizada
*/
void evaporate ( double *pheromones, int n_cities ); 

/* 
	reinforce()

	Atualiza a matriz de feromonios. 
	Para cada vertice da melhor solucao corrente, adiciona-se uma quantidade de feromonios. 

	Parametros: 
		pheromones: matriz de feromonios 
		distances: matriz de distancias 
		min_path: caminho minimo ( melhor solucao ) encontrado 
		n_cities: numero de cidades 

	Saida: 
		matriz de feromonios atualizada
*/
void reinforce ( double *pheromones, int *distances, int *min_path, int n_cities ); 

/* 
	run()

	Executa o algoritmo de colonia de formigas 

	Parametros: 	
		distances: matriz de distancias 
		n_cities: numero de cidades 
		n_ants: numero de formigas 

	Saida: 
		melhor entre as solucoes encontradas por todas as formigas 
*/
int *run ( int *distances, int n_cities, int n_ants ); 

__global__ void cuda_evaporate ( double *pheromones, int n_cities, double evap_rate );

__global__ void cuda_reinforce ( double *pheromones, int *distances, int *path, int n_cities, double amount );

__global__ void cuda_construct_tour (int *tours, int *visited, double *choiceinfo, double *probs, int n_cities );

int main ( int argc, char *argv[] ) {
	randomize();

	char const *inputname, *outputname;

	if ( argc < 2 ) {
		cout << "Missing input arguments!" << endl;
		cout << "Program " << argv[0] << " takes exactly 3 arguments." << endl;
		return 1;
	}

	if ( argc > 3 ) {
		cout << "Too many arguments in program " << argv[0] << "!" << endl;
		cout << "It takes exactly 3 arguments." << endl;
		return 1;
	}

	cout << "Running " << argv[0] << " with arguments: ";

	for (int i = 1; i < argc; i++) 
		cout << argv[i] << " ";
	cout << endl;

	inputname = argv[1];

	if ( !argv[2] ) {
		outputname = "results/output.txt";
	} else {
		outputname = argv[2];
	}

	int n_cities; /* Numero de cidades */ 	
	int *distances;	/* Matriz de distancias (distancia euclidiana) */ 
	
	/*
		Inicializa a instancia. 
		Executa o algoritmo e calcula do custo da solucao. 
	*/
	distances 	= load_instance ( inputname, n_cities );
	int *solution = run ( distances, n_cities, 128 ); 
	int cost = calculate_pathcost ( distances, solution, n_cities ); 

	cout << "Writing results in file " << outputname << "!\n";
	ofstream output;
	output.open(outputname); 

	output << "Custo: " << cost << endl; 
	output << "Melhor solucao encontrada:\n"; 
	for(int i=0; i<n_cities; i++) 
		output << solution[i] << endl;
	
	cout << argv[0] << " exited with no errors."; 

	return 0; 
}

__global__ void cuda_evaporate ( double *pheromones, int n_cities, double evap_rate ) { 

	int edge_id = threadIdx.x + blockIdx.x * blockDim.x; 
	pheromones[ edge_id ] *= evap_rate;
} 

__global__ void cuda_reinforce ( double *pheromones, int *distances, int *path, int n_cities, double amount ) {

	int col_id = threadIdx.x + blockIdx.x * blockDim.x;

	int origin = path[col_id];
	int dest = path[col_id+1];

	pheromones[ index( n_cities, origin, dest ) ] += amount;
	pheromones[ index( n_cities, dest, origin ) ] += amount; 
}

__global__ void cuda_construct_tour (int *tours, int *visited, double *choiceinfo, double *probs, int n_cities ) {

	//int line_id = blockIdx.x; 
	int line_id = blockDim.x * blockIdx.x + threadIdx.x; 
	
	//extern __shared__ int shared_visited[]; 

	//for(int i = 0; i < n_cities; i++) {
		//shared_visited[ index ( n_cities, line_id, i ) ] = visited[ index ( n_cities, line_id, i ) ];
	//} 

	// __syncthreads();
	
	for (int step = 1; step < n_cities; step++) { 

		int current = tours[ index ( n_cities, line_id, step - 1 ) ];
		double sum_probs = 0.0;

		for(int i = 0; i < n_cities; i++) {
			if ( visited[ index ( n_cities, line_id, i) ] == 1 ) 
				probs[ index ( n_cities, line_id, i ) ] = 0.0;
			else {
				double current_prob = choiceinfo[ index( n_cities, current, i ) ]; 
				probs[ index ( n_cities, line_id, i ) ] = current_prob; 
				sum_probs += current_prob; 
			}
		}

		double random;
		hiprandState_t state;
		hiprand_init ( (unsigned long long) clock(), 0, 0, &state );
		random = hiprand_uniform ( &state ); 
		random *= sum_probs; 

		int next;
		double sum = probs[ index ( n_cities, line_id, 0 ) ];

		for(next = 0; sum < random; next++) {
			sum += probs[ index ( n_cities, line_id, next + 1 ) ];
		} 

		tours[ index ( n_cities, line_id, step ) ] = next; 
		visited[ index ( n_cities, line_id, next) ] = 1; 
	} 	
}

int *load_instance ( char const *filename, int &n_cities ) {
	cout << "Opening file " << filename << endl;

	ifstream tsp; 
	tsp.open (filename); 

	/*if ( ifstream == NULL ) {
		cout << "File " << filename << " not found!\n";
		exit(1);
	}*/
	
	tsp >> n_cities; 
	
	int* distances = (int *) malloc ( n_cities * n_cities * sizeof(int) ); 
	
	for (int i = 0; i < n_cities; i++) 
		for (int j = 0; j < n_cities; j++) 
			tsp >> distances[ index(n_cities, i, j) ]; 
	
	return distances; 
}

int calculate_pathcost ( int *distances, int *path, int n_cities ) {
	int cost = 0; 
	
	for (int i = 0; i < (n_cities - 1); i++) 
		cost += distances[ index(n_cities, path[i], path[i+1]) ]; 
	return cost; 
}

int *best_solution ( int *tours, int *distances, int n_ants, int n_cities ) {
	int *best_tour = &tours[0];
	for (int tour = 0; tour < n_ants; tour++) 
		if (calculate_pathcost(distances, &tours[index(n_cities, tour, 0)], n_cities) < calculate_pathcost(distances, best_tour, n_cities)) 
			best_tour = &tours[index(n_cities, tour, 0)];
	return best_tour;
}

void evaporate ( double *pheromones, int n_cities ) { 

	int size = n_cities * n_cities * sizeof(double); 
	double *pheromones_device; 
	hipMalloc ( (void**) &pheromones_device, size); 
	
	hipMemcpy (pheromones_device, pheromones, size, hipMemcpyHostToDevice); 
	
	cuda_evaporate <<< n_cities, n_cities >>> ( pheromones_device, n_cities, EVAPORATION_RATE ); 
	
	hipMemcpy (pheromones, pheromones_device, size, hipMemcpyDeviceToHost); 
	
	hipFree (pheromones_device); 
}

void reinforce ( double *pheromones, int *distances, int *path, int n_cities ) { 
	double amount = (double) ( 1.0f / (double) calculate_pathcost ( distances, path, n_cities ) );

	int size_path = n_cities * sizeof(int);
	int size_int = n_cities * n_cities * sizeof(int);
	int size_double = n_cities * n_cities * sizeof(double);

	int *path_device;
	int *distances_device;
	double *pheromones_device;
	
	hipMalloc((void**)&path_device, size_path);
	hipMalloc((void**)&distances_device, size_int);
	hipMalloc((void**)&pheromones_device, size_double); 

	hipMemcpy (path_device, path, size_path, hipMemcpyHostToDevice);
	hipMemcpy (distances_device, distances, size_int, hipMemcpyHostToDevice); 
	hipMemcpy (pheromones_device, pheromones, size_double, hipMemcpyHostToDevice); 	

	cuda_reinforce <<< 1, n_cities - 1 >>> (pheromones_device, distances_device, path_device, n_cities, amount); 

	hipMemcpy (distances, distances_device, size_int, hipMemcpyDeviceToHost); 
	hipMemcpy (pheromones, pheromones_device, size_double, hipMemcpyDeviceToHost); 	
	
	hipFree (path_device); 
	hipFree (distances_device); 
	hipFree (pheromones_device); 
}

int *run ( int *distances, int n_cities, int n_ants) { 

	int ph_size = n_cities * n_cities * sizeof(double);
	int tours_size = n_ants * n_cities * sizeof(int);
	int dist_size = n_cities * n_cities * sizeof(int);

	double *pheromones = (double*) malloc ( ph_size ); 
	int *tours = (int*) malloc ( tours_size ); /* Solucoes */
	int *visited = (int*) malloc ( tours_size ); /* Lista de cidades visitadas */
	double *choiceinfo = (double*) malloc ( ph_size ); 

	int *distances_device; /* Copia da GPU da matriz de distancias */
	int *tours_device; /* Copia da GPU da matriz de solucoes */
	int *visited_device; /* Copia da GPU da matriz de cidades visitadas */ 
	double *choiceinfo_device; /* Copia da GPU da matriz de probabilidades (numeraodor) */
	double *probs; /* Matriz de probabilidades */

	hipMalloc ( (void**) &distances_device, dist_size ); 
	hipMalloc ( (void**) &tours_device, tours_size ); 
	hipMalloc ( (void**) &visited_device, tours_size ); 	
	hipMalloc ( (void**) &choiceinfo_device, ph_size ); 
	hipMalloc ( (void**) &probs, ph_size); 
		
	hipMemcpy ( distances_device, distances, dist_size, hipMemcpyHostToDevice );

	/*
		Instancia-se a matriz de feromonios. 
		Inicialmente, todas as arestas possuem a mesma quantidade de feromonios ( INIT_PHEROMONE_AMOUNT ). 
	*/ 
	for (int i = 0; i < n_cities; i++) 
		for (int j = 0; j < n_cities; j++) 
			pheromones[ index(n_cities, i, j) ] = INIT_PHEROMONE_AMOUNT; 

	for (int iteration = 0; iteration < NUMBER_OF_ITERATIONS; iteration++) { 
		/*
			Reseta todos os caminhos ao inicio de cada iteracao. 
			Inicialmente, todas as posicoes encontram-se no infinito. 
		*/

		for (int i = 0; i < n_ants; i++) 
			for (int j = 0; j < n_cities; j++) 
				tours[ index(n_cities, i, j) ] = Infinity; 

		for (int i = 0; i < n_ants; i++) 
			for (int j = 0; j < n_cities; j++)
				visited[ index(n_cities, i, j) ] = 0; 

		/*
			Calcula o numerador da funcao de probabilidade. 
			Em cada iteracao, este valor eh o mesmo para cada formiga, o que encoraja sua execucao aqui, 
			aumentando o desempenho do algoritmo. 
		*/
		
		for (int i = 0; i < n_cities; i++) {
			for (int j = 0; j < n_cities; j++) {
				double edge_pherom 	= pheromones[ index(n_cities, i, j) ]; 
				double edge_weight 	= distances[index(n_cities, i, j) ]; 
				double prob 		= 0.0f;
				if ( edge_weight != 0.0f ) {
					prob = pow ( edge_pherom, ALFA ) * pow ( (1/edge_weight), BETA ); 
				} else {
					prob = pow ( edge_pherom, ALFA ) * pow ( Infinity, BETA ); 
				} 
				choiceinfo[index(n_cities, i, j)] = prob; 
			} 
		} 		
		
		hipMemcpy ( choiceinfo_device, choiceinfo, ph_size, hipMemcpyHostToDevice ); 

		for (int ant = 0; ant < n_ants; ant++) { 
			int step = 0;

			/*
				Uma cidade inicial eh selecionada aleatoriamente. 
			*/
			int init = rand() % n_cities;

			/*
				Atualiza o tour ( para cada formiga ).
			*/
			tours [ index ( n_cities, ant, step ) ] = init;
			
			/*
				Atualiza a memoria da formiga. 	
			*/
			visited [ index ( n_cities, ant, init ) ] = 1; 
		} 

		hipMemcpy ( visited_device, visited, tours_size, hipMemcpyHostToDevice ); 
		hipMemcpy ( tours_device, tours, tours_size, hipMemcpyHostToDevice ); 

		int blockDim = 8;
		int antsPerBlock = n_ants / blockDim;
		// int sharedMemorySize = n_ants * n_cities * sizeof(int);
		
		cuda_construct_tour <<< blockDim, antsPerBlock/*, sharedMemorySize */>>> ( tours_device, visited_device, choiceinfo_device, probs, n_cities ); 

		hipMemcpy ( tours, tours_device, tours_size, hipMemcpyDeviceToHost ); 
		hipMemcpy ( visited, visited_device, tours_size, hipMemcpyDeviceToHost );	

		evaporate ( pheromones, n_cities ); 
		int *best = best_solution ( tours, distances, n_ants, n_cities ); 
		reinforce ( pheromones, distances, best, n_cities ); 
	} 

	hipFree ( distances_device ); 
	hipFree ( tours_device ); 
	hipFree ( visited_device ); 
	hipFree ( choiceinfo_device );	
	hipFree ( probs ); 
	
	int *best = best_solution ( tours, distances, n_ants, n_cities ); 
	return best; 
} 